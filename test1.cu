#include <stdio.h>
#include <stdlib.h>
#include <chrono>





void addTest(double *a_d, double *b_d, double *c_d, const int N{



	for(int t = 0; t < 10000000; ++t){
		for(int i = 0; i < N; ++i){
			c[i] += a[i] + b[i];
		}
	}
}


int main(){

	int N = 100;
	int time = 0;

	double *a_h, *b_h, *c_h;
	double *a_d, *b_d, *c_d;


	hipEvent_t tt1, tt2;
	float times;

	a_h = (double*)malloc(N * sizeof(double));
	b_h = (double*)malloc(N * sizeof(double));
	c_h = (double*)malloc(N * sizeof(double));


	hipMalloc((void **) &a_d, N * sizeof(double));
	hipMalloc((void **) &b_d, N * sizeof(double));
	hipMalloc((void **) &c_d, N * sizeof(double));

	for(int i = 0; i < N; ++i){
		a_h[i] = 0.1;
		b_h[i] = 100.0;
		c_h[i] = 0.0;
	}

	hipMemcpy(a_d, a_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, N * sizeof(double), hipMemcpyHostToDevice);


	hipEventCreate(&tt1);
	hipEventCreate(&tt2);

	hipEventRecord(tt1, 0);

	addTest_kernel <<< (N + 127) / 128, 128 >>>(a_d, b_d, c_d);

	hipEventRecord(tt2, 0);
	hipEventSynchronize(tt2);
	hipEventElapsedTime(&times, tt1, tt2);

	hipMemcpy(c_h, c_d, N * sizeof(double), hipMemcpyDeviceToHost);

	printf("Time in seconds:  %.8g\n", times * 0.001);

	//for(int i = 0; i < N; ++i){
	//	printf("%d %g\n", i, c_h[i]);
	//}


	return 0;
}
