
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>





__global__ void addTest_kernel(double *a_d, double *b_d, double *c_d, double t, const int N){

	int id = threadIdx.x * blockDim.x + blockIdx.x;

	if(id < N){
		c_d[id] += a_d[id] * t + 1.5 * t * b_d[id] + 0.8 * t * t;
	}
}


int main(){

	int N = 500000;

	double *a_h, *b_h, *c_h;
	double *a_d, *b_d, *c_d;


	hipEvent_t tt1, tt2;
	float times;

	a_h = (double*)malloc(N * sizeof(double));
	b_h = (double*)malloc(N * sizeof(double));
	c_h = (double*)malloc(N * sizeof(double));


	hipMalloc((void **) &a_d, N * sizeof(double));
	hipMalloc((void **) &b_d, N * sizeof(double));
	hipMalloc((void **) &c_d, N * sizeof(double));

	for(int i = 0; i < N; ++i){
		a_h[i] = 0.1;
		b_h[i] = 100.0;
		c_h[i] = 0.0;
	}

	hipMemcpy(a_d, a_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, N * sizeof(double), hipMemcpyHostToDevice);


	hipDeviceSynchronize();
	hipEventCreate(&tt1);
	hipEventCreate(&tt2);

	hipEventRecord(tt1, 0);

	addTest_kernel <<< (N + 127) / 128, 128 >>>(a_d, b_d, c_d, N, 1.5);

	hipEventRecord(tt2, 0);
	hipEventSynchronize(tt2);
	hipEventElapsedTime(&times, tt1, tt2);

	hipMemcpy(c_h, c_d, N * sizeof(double), hipMemcpyDeviceToHost);

	printf("Time in seconds:  %.8g\n", times * 0.001);

	//for(int i = 0; i < N; ++i){
	//	printf("%d %g\n", i, c_h[i]);
	//}


	return 0;
}
