
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>





__global__ void addTest_kernel(double *a_d, double *b_d, double *c_d, const int N){

	int id = threadIdx.x * blockDim.x + blockIdx.x;

	if(id < N){
		for(int t = 0; t < 10000000; ++t){
			c_d[id] += a_d[id] + b_d[id];
		}
	}
}


int main(){

	int N = 100;

	double *a_h, *b_h, *c_h;
	double *a_d, *b_d, *c_d;


	hipEvent_t tt1, tt2;
	float times;

	a_h = (double*)malloc(N * sizeof(double));
	b_h = (double*)malloc(N * sizeof(double));
	c_h = (double*)malloc(N * sizeof(double));


	hipMalloc((void **) &a_d, N * sizeof(double));
	hipMalloc((void **) &b_d, N * sizeof(double));
	hipMalloc((void **) &c_d, N * sizeof(double));

	for(int i = 0; i < N; ++i){
		a_h[i] = 0.1;
		b_h[i] = 100.0;
		c_h[i] = 0.0;
	}

	hipMemcpy(a_d, a_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, N * sizeof(double), hipMemcpyHostToDevice);


	hipEventCreate(&tt1);
	hipEventCreate(&tt2);

	hipEventRecord(tt1, 0);

	addTest_kernel <<< (N + 127) / 128, 128 >>>(a_d, b_d, c_d, N);

	hipEventRecord(tt2, 0);
	hipEventSynchronize(tt2);
	hipEventElapsedTime(&times, tt1, tt2);

	hipMemcpy(c_h, c_d, N * sizeof(double), hipMemcpyDeviceToHost);

	printf("Time in seconds:  %.8g\n", times * 0.001);

	//for(int i = 0; i < N; ++i){
	//	printf("%d %g\n", i, c_h[i]);
	//}


	return 0;
}
